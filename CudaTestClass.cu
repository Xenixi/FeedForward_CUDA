#include "hip/hip_runtime.h"
#include "CudaTestClass.cuh"
__global__ void createArrays(const int num, double *array0, double *array1){
    //initialize arrays
    int index = blockIdx.x * blockDim.x + threadIdx.x, stride = blockDim.x * gridDim.x;

    for(int i = index; i < num; i+=stride){
        array0[i] = 65.2314645;
        array1[i] = 56.5376367;
    }
}
__global__ void multiplyArrays(const int num, double *array0, double *array1){
    //matrix multiplication - result to array0
    int index = blockIdx.x * blockDim.x + threadIdx.x, stride = blockDim.x * gridDim.x;

    for(int i = index; i < num; i += stride){
        array0[i] = array0[i]*array1[i];
    }
}

int main(void){
    std::cout << "Running..." << std::endl;
    const int num = 25000000;
    double *array0, *array1;

    hipMallocManaged(&array0, sizeof(double)*num);
    hipMallocManaged(&array1, sizeof(double)*num);

    //ignore errors from Intellisense / launch params
    #ifndef __INTELLISENSE__
    createArrays<<<9766,1024>>>(num, array0, array1);
    #endif

    hipDeviceSynchronize();

    #ifndef __INTELLISENSE__
    multiplyArrays<<<9766,1024>>>(num, array0, array1);
    #endif

    hipDeviceSynchronize();

    hipFree(array0);
    hipFree(array1);
    
    return 0;
}
