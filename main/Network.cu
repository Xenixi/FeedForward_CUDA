#include <iostream>
#include <math.h>
#include <fstream>
#include <sstream>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <vector>
#include "Network.cuh"

namespace anncuda0
{
    struct NetworkBackbone
    {
        struct NodeParams
        {
            int iNodes, oNodes, hNodes;
            int iter;
        };
        NodeParams np;
        NetworkBackbone(int iNodes, int oNodes, int hNodes)
        {
            np.iNodes = iNodes;
            np.oNodes = oNodes;
            np.hNodes = hNodes;
        }

        //error " error: invalid redeclaration of type name "Network"
        //D:\REPOLOCAL\Cloned\parallel-computing\FeedForward_CUDA\main\Network.cuh(13): here"
        //---
        void train(float *inputs, float *targets)
        {
        }
        void query(float *inputs)
        {
        }
        int getInputQuantity()
        {
            return np.iNodes;
        }
        int getOutputQuantity()
        {
            return np.oNodes;
        }
        int getHiddenQuantity()
        {
            return np.hNodes;
        }
    };
} // namespace anncuda0

int main(void)
{

    std::cout << "hello" << std::endl;
    anncuda0::NetworkBackbone n(/*Input*/ 784, /*Output*/ 10, /*Hidden*/ 200);

    float *inputs, *targets;
    hipMallocManaged(&inputs, sizeof(float) * n.getInputQuantity());
    hipMallocManaged(&targets, sizeof(float) * n.getInputQuantity());

    std::cout << "Reading inputs..." << std::endl;

    std::ifstream inputsFile("dat/trainMNIST.csv");
    std::stringstream bufferStream;
    bufferStream << inputsFile.rdbuf();
    std::string str;

    char delim = ',';

    std::vector<std::string> strings;
    for (int i = 0; std::getline(bufferStream, str, delim); i++)
    {
        strings.push_back(str);
    }

    for (std::string str2 : strings)
    {
        std::cout << "Read: " << str2 << std::endl;
    }

    hipFree(inputs);
    hipFree(targets);

    return 0;
}
