#include <iostream>
#include <chrono>
#include <math.h>
#include <fstream>
#include <sstream>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <vector>
#include "NetworkBackbone.cuh"

int main(void)
{

    std::cout << "hello" << std::endl;
    NetworkBackbone n(/*Input*/ 784, /*Output*/ 10, /*Hidden*/ 200);

    float *inputs, *targets;
    hipMallocManaged(&inputs, sizeof(float) * n.getInputQuantity());
    hipMallocManaged(&targets, sizeof(float) * n.getInputQuantity());

    std::cout << "Reading inputs..." << std::endl;

    auto time1 = std::chrono::high_resolution_clock::now();

    std::ifstream inputsFile("dat/mnist_train.csv");
    std::stringstream bufferStream;
    bufferStream << inputsFile.rdbuf();
    std::string str;

    char delim = ',';

    std::vector<std::string> trainStrings;
    int i;
    for (i = 0; std::getline(bufferStream, str, delim); i++)
    {
        trainStrings.push_back(str);
    }
    std::cout << "Train - Total: " << i << std::endl;
    //test file

    std::ifstream testFile("dat/mnist_test.csv");
    std::stringstream bufferStream2;
    bufferStream2 << testFile.rdbuf();
    std::string str2;

    std::vector<std::string> testStrings;

    for (i = 0; std::getline(bufferStream2, str2, delim); i++)
    {
        testStrings.push_back(str2);
    }

    std::cout << "Test - Total: " << i << std::endl;
   
    auto time2 = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(time2 - time1).count();
    std::cout << "Completed.\nTime taken: " << duration << "ms" << std::endl;

    hipFree(inputs);
    hipFree(targets);



    return 0;
}
