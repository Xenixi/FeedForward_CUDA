#include "hip/hip_runtime.h"
//#include "Network.cuh"
#include <iostream>
#include <math.h>
#include "NetworkBackbone.cuh"
//runs on GPU
__global__ void initWeights(float *weightsInputHidden, float *weightsHiddenOutput, int iNodes, int hNodes, int oNodes){
    int idx = blockDim.x * blockIdx.x + threadIdx.x, stride = blockDim.x * gridDim.x;
    
    for(int i = idx; i < hNodes*iNodes; i+=stride){

    } 
    for(int i = idx; i < hNodes*oNodes; i+=stride){

    }
}

__global__ void trainNetwork(float *inputs, float *targets, int iNodes, int hNodes, int oNodes)
{


}

__global__ void queryNetwork(float *inputs)
{
}

__device__ float activation(float input)
{
    return input / (abs(input) + 1);
}

/////////////////////
int *NetworkBackbone::getDeviceProperties()
{
    Utils u;
    int *props = new int[3];
    props[0] = u.getSMs();
    props[1] = u.getTB();
    props[2] = u.getTMP();

    return props;
}
int NetworkBackbone::Utils::getSMs()
{
    return getDeviceProps().multiProcessorCount;
}

int NetworkBackbone::Utils::getTB()
{
    return getDeviceProps().maxThreadsPerBlock;
}
int NetworkBackbone::Utils::getTMP()
{
    return getDeviceProps().maxThreadsPerMultiProcessor;
}

hipDeviceProp_t NetworkBackbone::Utils::getDeviceProps()
{
    int device;
    hipGetDevice(&device);
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);

    return properties;
}
//CPU
//continued
NetworkBackbone::NodeParams np;
NetworkBackbone::NetworkBackbone(int iNodes, int oNodes, int hNodes)
{
    np.iNodes = iNodes;
    np.oNodes = oNodes;
    np.hNodes = hNodes;

    //initialization


}

void NetworkBackbone::train(float *inputs, float *targets)
{
    //allocate shared memory GPU
    //Maybe move ALL values to GPU ahead of time?
    hipMallocManaged(&inputs, sizeof(float) * getInputQuantity());
    hipMallocManaged(&targets, sizeof(float) * getOutputQuantity());

    //kernel call
    int blocks = (getDeviceProperties()[0] * getDeviceProperties()[2] / getDeviceProperties()[1]);
    int blockThreads = getDeviceProperties()[1];

    trainNetwork<<<blocks, blockThreads>>>(inputs, targets, getInputQuantity(), getHiddenQuantity(), getOutputQuantity());

    //free

    hipFree(inputs);
    hipFree(targets);
}
void NetworkBackbone::query(float *inputs)
{
}
int NetworkBackbone::getInputQuantity()
{
    return np.iNodes;
}
int NetworkBackbone::getOutputQuantity()
{
    return np.oNodes;
}
int NetworkBackbone::getHiddenQuantity()
{
    return np.hNodes;
}
