#include "hip/hip_runtime.h"
//#include "Network.cuh"
#include <iostream>
#include <math.h>
#include <random>
#include "NetworkBackbone.cuh"

// https://youtu.be/5IkodnY0PeY?t=592
//runs on GPU
/*__global__ void initWeights(float *weightsInputHidden, float *weightsHiddenOutput, int iNodes, int hNodes, int oNodes){
    int idx = blockDim.x * blockIdx.x + threadIdx.x, stride = blockDim.x * gridDim.x;
    
    for(int i = idx; i < hNodes*iNodes; i+=stride){
        
    } 
    for(int i = idx; i < hNodes*oNodes; i+=stride){

    }
}
*/
__global__ void trainNetwork(float *inputs, float *targets, int iNodes, int hNodes, int oNodes)
{
}

__global__ void queryNetwork(float *inputs, float *weightsInputHidden, float *weightsHiddenOutput, int iNodes, int hNodes, int oNodes)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x, stride = blockDim.x * gridDim.x;
    //CONTINUE HERE PG 132 PYTHON BOOK & http://luniak.io/cuda-neural-network-implementation-part-1/#implementation-plan
    //
    for (int i = idx; i < iNodes; i += stride)
    {
        inputs[i] = inputs[i] * weightsInputHidden[i];
        inputs[i] = activation(inputs[i]);
        inputs[i] = inputs[i] * weightsHiddenOutput[i];
        inputs[i] = activation(inputs[i]);
    }
}

__device__ float activation(float input)
{
    return input / (abs(input) + 1);
}

/////////////////////
int *NetworkBackbone::getDeviceProperties()
{
    Utils u;
    int *props = new int[3];
    props[0] = u.getSMs();
    props[1] = u.getTB();
    props[2] = u.getTMP();

    return props;
}
int NetworkBackbone::Utils::getSMs()
{
    return getDeviceProps().multiProcessorCount;
}

int NetworkBackbone::Utils::getTB()
{
    return getDeviceProps().maxThreadsPerBlock;
}
int NetworkBackbone::Utils::getTMP()
{
    return getDeviceProps().maxThreadsPerMultiProcessor;
}

hipDeviceProp_t NetworkBackbone::Utils::getDeviceProps()
{
    int device;
    hipGetDevice(&device);
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);

    return properties;
}
//CPU
//continued
NetworkBackbone::NodeParams np;
NetworkBackbone::NetworkBackbone(int iNodes, int oNodes, int hNodes)
{
    np.iNodes = iNodes;
    np.oNodes = oNodes;
    np.hNodes = hNodes;

    //initialization
    float *weightsInputsHidden, *weightsHiddenOutput;

    hipMallocManaged(&weightsInputsHidden, sizeof(float) * np.iNodes * np.hNodes);
    hipMallocManaged(&weightsHiddenOutput, sizeof(float) * np.hNodes * np.oNodes);

    init(weightsInputsHidden, weightsHiddenOutput, np.iNodes, np.hNodes, np.oNodes);
}
/****************
intiialization
****************/
void NetworkBackbone::init(float *weightsInputHidden, float *weightsHiddenOutput, int iNodes, int hNodes, int oNodes)
{
    for (int i = 0; i < (iNodes * hNodes); i++)
    {
        std::random_device rnd;
        std::mt19937 mt1(rnd());
        std::uniform_real_distribution<float> dist(-0.5, 0.5);

        weightsInputHidden[i] = dist(mt1);
        std::cout << "val_" << i << ": " << weightsInputHidden[i] << std::endl;
    }
}

void NetworkBackbone::train(float *inputs, float *targets)
{
    //allocate shared memory GPU
    //Maybe move ALL values to GPU ahead of time?
    hipMallocManaged(&inputs, sizeof(float) * getInputQuantity());
    hipMallocManaged(&targets, sizeof(float) * getOutputQuantity());

    //kernel call
    int blocks = (getDeviceProperties()[0] * getDeviceProperties()[2] / getDeviceProperties()[1]);
    int blockThreads = getDeviceProperties()[1];

    trainNetwork<<<blocks, blockThreads>>>(inputs, targets, getInputQuantity(), getHiddenQuantity(), getOutputQuantity());

    //free

    hipFree(inputs);
    hipFree(targets);
}
void NetworkBackbone::query(float *inputs, float *weightsInputHidden, float *weightsHiddenOutput, int iNodes, int hNodes, int oNodes)
{
    int blocks = (getDeviceProperties()[0] * getDeviceProperties()[2] / getDeviceProperties()[1]);
    int blockThreads = getDeviceProperties()[1];

    queryNetwork<<<blocks, blockThreads>>>(inputs, weightsInputHidden, weightsHiddenOutput, iNodes, hNodes, oNodes);
}
int NetworkBackbone::getInputQuantity()
{
    return np.iNodes;
}
int NetworkBackbone::getOutputQuantity()
{
    return np.oNodes;
}
int NetworkBackbone::getHiddenQuantity()
{
    return np.hNodes;
}
