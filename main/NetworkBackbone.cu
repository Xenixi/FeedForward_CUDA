#include "hip/hip_runtime.h"
//#include "Network.cuh"
#include <iostream>
#include <math.h>
#include <random>
#include "NetworkBackbone.cuh"

// https://youtu.be/5IkodnY0PeY?t=592
//runs on GPU
/*__global__ void initWeights(float *weightsInputHidden, float *weightsHiddenOutput, int iNodes, int hNodes, int oNodes){
    int idx = blockDim.x * blockIdx.x + threadIdx.x, stride = blockDim.x * gridDim.x;
    
    for(int i = idx; i < hNodes*iNodes; i+=stride){
        
    } 
    for(int i = idx; i < hNodes*oNodes; i+=stride){

    }
}
*/
__global__ void trainNetwork(float *inputs, float *targets, float *weightsInputHidden, float *weightsHiddenOutput, int iNodes, int hNodes, int oNodes, float *learningRate, float *outputErrs, float *finalOutputs, float *hiddenOutputs, float *hiddenErrs)
{
    //  THE INPUT VARIABLE PASSED IN WILL BE REPLACED WITH THE RETURNED ERROR VALUES SO MAKE A COPY OF THE VARIABLE!!!!
    
    //does this work in CUDA GPU kernel function?...
    float *inputsOrig = inputs;



    int idx = blockDim.x * blockIdx.x + threadIdx.x, stride = blockDim.x * gridDim.x;

    for (int i = idx; i < iNodes; i += stride)
    {
        inputs[i] = inputs[i] * weightsInputHidden[i];
        hiddenOutputs[i] = activation(inputs[i]);

    /// final_inputs = numpy.dot(self.who, hidden_outputs)
        inputs[i] = weightsHiddenOutput[i] * hiddenOutputs[i];
    /// final_outputs = self.activation_function(final_inputs)
        finalOutputs[i] = activation(inputs[i]);
    ///output_errors = targets - final_outputs
        outputErrs[i] = targets[i] - finalOutputs[i];
    ///hidden_errors = numpy.dot(self.who.T, output_errors)
        hiddenErrs[i] = weightsHiddenOutput[i] * outputErrs[i];
    //// self.who += self.lr * numpy.dot((output_errors * final_outputs * (1.0 - final_outputs)), numpy.transpose(hidden_outputs))
    //// self.wih += self.lr * numpy.dot((hidden_errors * hidden_outputs * (1.0 - hidden_outputs)), numpy.transpose(inputs))
    //**************************************************************************
    //  IF SOMETHING GOES WRONG IT'S PROBABLY THIS THIS IS PROBABLY BROKEN MAY NEED TO BE FIXED- THE PY ONE USED NUMPY TRANSPOSE AND 2D ARRAYS THIS DOESN'T
       weightsHiddenOutput[i] += (learningRate * (outputErrs[i]*finalOutputs[i]*(1.0-finalOutputs[i])) * hiddenOutputs[i]);
       weightsInputHidden[i] += (learningRate * (hiddenErrs[i]*hiddenOutputs[i]*(1.0-hiddenOutputs[i])) * inputsOrig[i]);

        ///LINE 66 LEFT OFF (PYTHON PROGRAM)
        ///NUMPY.TRANSPOSE LINES AND STUFF FROM THE PYTHON (LINE 71 IN THE OTHER PROGRAM - )
    }
}

__global__ void queryNetwork(float *inputs, float *weightsInputHidden, float *weightsHiddenOutput, int iNodes, int hNodes, int oNodes)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x, stride = blockDim.x * gridDim.x;
    //CONTINUE HERE PG 132 PYTHON BOOK & http://luniak.io/cuda-neural-network-implementation-part-1/#implementation-plan
    //
    for (int i = idx; i < iNodes; i += stride)
    {
        inputs[i] = inputs[i] * weightsInputHidden[i];
        inputs[i] = activation(inputs[i]);
        inputs[i] = inputs[i] * weightsHiddenOutput[i];
        inputs[i] = activation(inputs[i]);
    }
}

__device__ float activation(float input)
{
    return input / (abs(input) + 1);
}

/////////////////////
int *NetworkBackbone::getDeviceProperties()
{
    Utils u;
    int *props = new int[3];
    props[0] = u.getSMs();
    props[1] = u.getTB();
    props[2] = u.getTMP();

    return props;
}
int NetworkBackbone::Utils::getSMs()
{
    return getDeviceProps().multiProcessorCount;
}

int NetworkBackbone::Utils::getTB()
{
    return getDeviceProps().maxThreadsPerBlock;
}
int NetworkBackbone::Utils::getTMP()
{
    return getDeviceProps().maxThreadsPerMultiProcessor;
}

hipDeviceProp_t NetworkBackbone::Utils::getDeviceProps()
{
    int device;
    hipGetDevice(&device);
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);

    return properties;
}
//CPU
//continued
NetworkBackbone::NodeParams np;
NetworkBackbone::NetworkBackbone(int iNodes, int oNodes, int hNodes)
{
    np.iNodes = iNodes;
    np.oNodes = oNodes;
    np.hNodes = hNodes;

    //initialization
    float *weightsInputsHidden, *weightsHiddenOutput;

    hipMallocManaged(&weightsInputsHidden, sizeof(float) * np.iNodes * np.hNodes);
    hipMallocManaged(&weightsHiddenOutput, sizeof(float) * np.hNodes * np.oNodes);

    init(weightsInputsHidden, weightsHiddenOutput, np.iNodes, np.hNodes, np.oNodes);
}
/****************
intiialization
****************/
void NetworkBackbone::init(float *weightsInputHidden, float *weightsHiddenOutput, int iNodes, int hNodes, int oNodes)
{
    for (int i = 0; i < (iNodes * hNodes); i++)
    {
        std::random_device rnd;
        std::mt19937 mt1(rnd());
        std::uniform_real_distribution<float> dist(-0.5, 0.5);

        weightsInputHidden[i] = dist(mt1);
        std::cout << "val_" << i << ": " << weightsInputHidden[i] << std::endl;
    }
}

void NetworkBackbone::train(float *inputs, float *targets, float *learningRate, float *outputErrs, float *finalOutputs, float *hiddenOutputs, float *hiddenErrs)
{
    //allocate shared memory GPU
    //Maybe move ALL values to GPU ahead of time?
    hipMallocManaged(&inputs, sizeof(float) * getInputQuantity());
    hipMallocManaged(&targets, sizeof(float) * getOutputQuantity());

    //only need 1 for the learning rate (not an array-like type)
    hipMallocManaged(&learningRate, sizeof(float));

    hipMallocManaged(&outputErrs, sizeof(float) * getOutputQuantity());
    hipMallocManaged(&finalOutputs, sizeof(float) * getOutputQuantity());
    hipMallocManaged(&hiddenOutputs, sizeof(float) * getHiddenQuantity());
    hipMallocManaged(&hiddenErrs, sizeof(float) * getHiddenQuantity());

    //kernel call
    int blocks = (getDeviceProperties()[0] * getDeviceProperties()[2] / getDeviceProperties()[1]);
    int blockThreads = getDeviceProperties()[1];

    trainNetwork<<<blocks, blockThreads>>>(inputs, targets, getInputQuantity(), getHiddenQuantity(), getOutputQuantity(), learningRate, outputErrs, finalOutputs, hiddenOutputs, hiddenErrs);

    //free

    hipFree(inputs);
    hipFree(targets);
}
void NetworkBackbone::query(float *inputs, float *weightsInputHidden, float *weightsHiddenOutput, int iNodes, int hNodes, int oNodes)
{
    int blocks = (getDeviceProperties()[0] * getDeviceProperties()[2] / getDeviceProperties()[1]);
    int blockThreads = getDeviceProperties()[1];

    queryNetwork<<<blocks, blockThreads>>>(inputs, weightsInputHidden, weightsHiddenOutput, iNodes, hNodes, oNodes);
}
int NetworkBackbone::getInputQuantity()
{
    return np.iNodes;
}
int NetworkBackbone::getOutputQuantity()
{
    return np.oNodes;
}
int NetworkBackbone::getHiddenQuantity()
{
    return np.hNodes;
}
