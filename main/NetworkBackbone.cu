//#include "Network.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "NetworkBackbone.cuh"

NetworkBackbone::NodeParams np;
NetworkBackbone::NetworkBackbone(int iNodes, int oNodes, int hNodes)
{
    np.iNodes = iNodes;
    np.oNodes = oNodes;
    np.hNodes = hNodes;
}


void NetworkBackbone::train(float *inputs, float *targets)
{
    
}
void NetworkBackbone::query(float *inputs)
{
}
int NetworkBackbone::getInputQuantity()
{
    return np.iNodes;
}
int NetworkBackbone::getOutputQuantity()
{
    return np.oNodes;
}
int NetworkBackbone::getHiddenQuantity()
{
    return np.hNodes;
}
