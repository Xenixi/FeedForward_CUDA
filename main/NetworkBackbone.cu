#include "hip/hip_runtime.h"
//#include "Network.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "NetworkBackbone.cuh"

__global__ void trainNetwork(float *inputs, float *targets, int iNodes, int hNodes, int oNodes)
{
}

__global__ void queryNetwork(float *inputs)
{
}

NetworkBackbone::NodeParams np;
NetworkBackbone::NetworkBackbone(int iNodes, int oNodes, int hNodes)
{
    np.iNodes = iNodes;
    np.oNodes = oNodes;
    np.hNodes = hNodes;
}


void NetworkBackbone::train(float *inputs, float *targets)
{
    //allocate shared memory GPU
    //Maybe move ALL values to GPU ahead of time?
    hipMallocManaged(&inputs, sizeof(float) * getInputQuantity());
    hipMallocManaged(&targets, sizeof(float) * getOutputQuantity());

    //kernel call
    trainNetwork<<<8192, 1024>>>(inputs, targets, getInputQuantity(), getHiddenQuantity(), getOutputQuantity());

    //free

    hipFree(inputs);
    hipFree(targets);
}
void NetworkBackbone::query(float *inputs)
{
}
int NetworkBackbone::getInputQuantity()
{
    return np.iNodes;
}
int NetworkBackbone::getOutputQuantity()
{
    return np.oNodes;
}
int NetworkBackbone::getHiddenQuantity()
{
    return np.hNodes;
}
